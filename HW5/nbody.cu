#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------------- 
// CUDA code to compute minimun distance between n points
//
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <limits>
#include <float.h>

#define MAX_POINTS 1048576
#define block_size 1024
// ---------------------------------------------------------------------------- 
// Kernel Function to compute distance between all pairs of points
// Input: 
//	X: X[i] = x-coordinate of the ith point
//	Y: Y[i] = y-coordinate of the ith point
//	n: number of points
// Output: 
//	D: D[0] = minimum distance
//
__device__ unsigned int blocks_done = 0;
__global__ void minimum_distance(float * X, float * Y, volatile float * D, int n) {
__shared__ float block[block_size];
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    float dx, dy, local_minimum;
    int j, k;
    float minimum = FLT_MAX;
    bool isLastBlockDone;
    
    for(i = 0; i < n - 1; i++) {
    for(j = i + 1; j < n; j++){
        dx = X[j] - X[i];
        dy = Y[j] - Y[i];
        local_minimum = sqrtf(dx * dx + dy * dy);
        if(local_minimum < minimum){
            minimum = local_minimum;
        }
    }
    block[threadIdx.x] = minimum;
    __syncthreads();
    int block_end = (n % block_size);
    if(block_end == 0){
        block_end = block_size;
    }
    else if(blockIdx.x != n/block_size){
        block_end = block_size;
    }
    }
    for(k = 1; k < block_end; k *= 2) {
        if(threadIdx.x % (2 * k) == 0 && (threadIdx.x + k) < block_end - 1){
        if(block[threadIdx.x] > block[threadIdx.x + k]){
                block[threadIdx.x] = block[threadIdx.x + k];
            }
        __syncthreads();
        }
    }
    if(threadIdx.x == 0) {
        D[blockIdx.x] = block[0];
        int value = atomicInc(&blocks_done, gridDim.x);
        isLastBlockDone = (value == (gridDim.x - 1));
    }
    if(isLastBlockDone && threadIdx.x == 0) {
        int blocks = n / block_size + (n % block_size != 0);
        for(k = 1; k < blocks; k++){
            if(D[0] > D[k]) {
                D[0] = D[k];            // Global Min
            }
        }
    }
}

// ---------------------------------------------------------------------------- 
// Host function to compute minimum distance between points
// Input:
//	X: X[i] = x-coordinate of the ith point
//	Y: Y[i] = y-coordinate of the ith point
//	n: number of points
// Output: 
//	D: minimum distance
//
float minimum_distance_host(float * X, float * Y, int n) {
    float dx, dy, Dij, min_distance, min_distance_i;
    int i, j;
    dx = X[1]-X[0];
    dy = Y[1]-Y[0];
    min_distance = sqrtf(dx*dx+dy*dy);
    for (i = 0; i < n-1; i++) {
	for (j = i+1; j < i+2; j++) {
	    dx = X[j]-X[i];
	    dy = Y[j]-Y[i];
	    min_distance_i = sqrtf(dx*dx+dy*dy);
	}
	for (j = i+1; j < n; j++) {
	    dx = X[j]-X[i];
	    dy = Y[j]-Y[i];
	    Dij = sqrtf(dx*dx+dy*dy);
	    if (min_distance_i > Dij) min_distance_i = Dij;
	}
	if (min_distance > min_distance_i) min_distance = min_distance_i;
    }
    return min_distance;
}
// ---------------------------------------------------------------------------- 
// Print device properties
void print_device_properties() {
    int i, deviceCount;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount(&deviceCount);
    printf("------------------------------------------------------------\n");
    printf("Number of GPU devices found = %d\n", deviceCount);
    for ( i = 0; i < deviceCount; ++i ) {
	hipGetDeviceProperties(&deviceProp, i);
	printf("[Device: %1d] Compute Capability %d.%d.\n", i, deviceProp.major, deviceProp.minor);
	printf(" ... multiprocessor count  = %d\n", deviceProp.multiProcessorCount); 
	printf(" ... max threads per multiprocessor = %d\n", deviceProp.maxThreadsPerMultiProcessor); 
	printf(" ... max threads per block = %d\n", deviceProp.maxThreadsPerBlock); 
	printf(" ... max block dimension   = %d, %d, %d (along x, y, z)\n",
		deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]); 
	printf(" ... max grid size         = %d, %d, %d (along x, y, z)\n",
		deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]); 
	printf(" ... warp size             = %d\n", deviceProp.warpSize); 
	printf(" ... clock rate            = %d MHz\n", deviceProp.clockRate/1000); 
    }
    printf("------------------------------------------------------------\n");
}
// ---------------------------------------------------------------------------- 
// Main program - initializes points and computes minimum distance 
// between the points
//
int main(int argc, char* argv[]) {

    // Host Data
    float * hVx;		// host x-coordinate array
    float * hVy;		// host y-coordinate array
    float * hmin_dist;		// minimum value on host

    // Device Data
    float * dVx;		// device x-coordinate array
    float * dVy;		// device x-coordinate array
    float * dmin_dist;		// minimum value on device

    // Device parameters
    int MAX_BLOCK_SIZE;
    
    // Timing variables
    hipEvent_t start, stop;		// GPU timing variables
    struct timespec cpu_start, cpu_stop; // CPU timing variables
    float time_array[10]; 

    // Other variables
    int i, size, num_points; 
    float min_distance, sqrtn;
    int seed = 0;

    // Print device properties
    print_device_properties(); 

    // Get device information and set device to use
    int deviceCount;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount > 0) {
	hipSetDevice(0); 
	hipGetDeviceProperties(&deviceProp, 0);
	MAX_BLOCK_SIZE = deviceProp.maxThreadsPerBlock;
    } else {
	printf("Warning: No GPU device found ... results may be incorrect\n");
    }

    // Timing initializations
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Check input
    if (argc != 2) {
	printf("Use: %s <number of points>\n", argv[0]);  
	exit(0);
    }
    if ((num_points = atoi(argv[argc-1])) < 2) {
	printf("Minimum number of points allowed: 2\n");
	exit(0);
    } 
    if ((num_points = atoi(argv[argc-1])) > MAX_POINTS) {
	printf("Maximum number of points allowed: %d\n", MAX_POINTS);
	exit(0);
    } 

    // Allocate host coordinate arrays 
    size = num_points * sizeof(float); 
    hVx = (float *) malloc(size); 
    hVy = (float *) malloc(size);
    hmin_dist = (float *) malloc(size);

    // Initialize points
    srand48(seed);
    sqrtn = (float) sqrt(num_points); 
    for (i = 0; i < num_points; i++) {
	hVx[i] = sqrtn * (float)drand48();
	hVy[i] = sqrtn * (float)drand48();
    }

    // Allocate device coordinate arrays
    hipMalloc(&dVx, size);
    hipMalloc(&dVy, size);
    hipMalloc(&dmin_dist, size);

    // Copy coordinate arrays from host memory to device memory 
    hipEventRecord( start, 0 ); 

    hipMemcpy(dVx, hVx, size, hipMemcpyHostToDevice);
    hipMemcpy(dVy, hVy, size, hipMemcpyHostToDevice);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&(time_array[0]), start, stop);

    // Invoke kernel
    hipEventRecord( start, 0 ); 

    // Custom variables to create blocks
    int blocks = num_points / (block_size) + ((num_points % (block_size)) != 0);

    // Kernel function invocation
    minimum_distance<<<blocks, block_size>>>(dVx, dVy, dmin_dist, num_points);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&(time_array[1]), start, stop);

    // Copy result from device memory to host memory 
    hipEventRecord( start, 0 ); 

    hipMemcpy(hmin_dist, dmin_dist, sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&(time_array[2]), start, stop);

    // Compute minimum distance on host to check device computation
    clock_gettime(CLOCK_REALTIME, &cpu_start);

    min_distance = minimum_distance_host(hVx, hVy, num_points); 

    clock_gettime(CLOCK_REALTIME, &cpu_stop);
    time_array[3] = 1000*((cpu_stop.tv_sec-cpu_start.tv_sec)                    
	    +0.000000001*(cpu_stop.tv_nsec-cpu_start.tv_nsec));

    // Print results
    printf("Number of Points    = %d\n", num_points); 
    printf("GPU Host-to-device  = %f ms \n", time_array[0]);
    printf("GPU Device-to-host  = %f ms \n", time_array[2]);
    printf("GPU execution time  = %f ms \n", time_array[1]);
    printf("CPU execution time  = %f ms\n", time_array[3]);
    printf("Min. distance (GPU) = %e\n", hmin_dist[0]);
    printf("Min. distance (CPU) = %e\n", min_distance);
    printf("Relative error      = %e\n", fabs(min_distance-hmin_dist[0])/min_distance);

    // Free device memory 
    hipFree(dVx);
    hipFree(dVy);
    hipFree(dmin_dist);

    // Free host memory 
    free(hVx);
    free(hVy);
}  
